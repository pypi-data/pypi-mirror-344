
#include <hip/hip_runtime.h>
extern "C" __global__
void local_maxima(const float* image, float threshold, int delta, int delta_fit,
				  unsigned short* z_out, unsigned short* x_out, unsigned short* y_out,
				  unsigned int* count,
				  int depth, int height, int width, int max_points) {
	// Get flattened index
	int idx = blockDim.x * blockIdx.x + threadIdx.x;
	if (idx >= depth * height * width) {
		return;
	}

	// Convert flat index to 3D coordinates
	int z = idx / (height * width);
	int temp = idx % (height * width);
	int x = temp / width;
	int y = temp % width;


	// Check if above threshold
	if (image[idx] <= threshold) {
		return;
	}

	// Check if it's a local maximum in the neighborhood
	bool is_max = true;
	for (int dz = -delta; dz <= delta; dz++) {
		for (int dx = -delta; dx <= delta; dx++) {
			for (int dy = -delta; dy <= delta; dy++) {
				// Skip the center point
				if (dz == 0 && dx == 0 && dy == 0) {
					continue;
				}

				// Check if within spherical mask
				if ((dz*dz + dx*dx + dy*dy) > (delta*delta)) {
					continue;
				}

				int nz = z + dz;
				int nx = x + dx;
				int ny = y + dy;
				

				// Apply reflect only if out of bounds
				if (nz < 0 || nz >= depth) {
					nz = (nz < 0) ? -nz : 2 * depth - nz - 2;
				}
				if (nx < 0 || nx >= height) {
					nx = (nx < 0) ? -nx : 2 * height - nx - 2;
				}
				if (ny < 0 || ny >= width) {
					ny = (ny < 0) ? -ny : 2 * width - ny - 2;
				}

				if (image[idx] < image[nz * height * width + nx * width + ny]) {
					is_max = false;
					break;
				}
			}
			if (!is_max) break;
		}
		if (!is_max) break;
	}

	if (is_max) {
		// If it's a local maximum, add to output
		unsigned int pos = atomicAdd(count, 1);
		if (pos < max_points) {
		   	z_out[pos] = z;
		   	x_out[pos] = x;
		   	y_out[pos] = y;
		}
	}
}

#define MAX_KERNEL_POINTS 515
extern "C" __global__
void delta_fit_cross_corr(
	const float* __restrict__ image,
	const unsigned short* __restrict__ raw,
	unsigned short* __restrict__ z_out,   // (num_maxima)
	unsigned short* __restrict__ x_out,   // (num_maxima)
	unsigned short* __restrict__ y_out,   // (num_maxima)
	float* __restrict__ output,		// (num_maxima, 8) [zc, xc, yc, background, habs, h, cross_corr, delta_fit_value]
	int num_maxima,
	int Z, int X, int Y,
	int delta_fit,
	float sigmaZ, float sigmaXY
) {
	int idx = blockIdx.x * blockDim.x + threadIdx.x;
	if (idx >= num_maxima) return;

	int z0 = z_out[idx];
	int x0 = x_out[idx];
	int y0 = y_out[idx];

	float sum_val = 0.0f;
	float sum_z = 0.0f;
	float sum_x = 0.0f;
	float sum_y = 0.0f;
	float min_val = 1e20f;

	// Cross-correlation computation (Gaussian weights)
	float norm_G[MAX_KERNEL_POINTS];
	float sample_vals[MAX_KERNEL_POINTS];
	float raw_vals[MAX_KERNEL_POINTS];
	int count = 0;

	// Step 1: Collect all samples and calculate Gaussian weights
	for (int dz = -delta_fit; dz <= delta_fit; ++dz) {
		for (int dx = -delta_fit; dx <= delta_fit; ++dx) {
			for (int dy = -delta_fit; dy <= delta_fit; ++dy) {
			    if (dz * dz + dx * dx + dy * dy > delta_fit * delta_fit) continue;
			    if (count >= 125) continue;  // Safety check

			    int zz = z0 + dz;
			    int xx = x0 + dx;
			    int yy = y0 + dy;

			    // Reflect if out of bounds
			    zz = zz < 0 ? -zz : (zz >= Z ? 2 * Z - zz - 2 : zz);
			    xx = xx < 0 ? -xx : (xx >= X ? 2 * X - xx - 2 : xx);
			    yy = yy < 0 ? -yy : (yy >= Y ? 2 * Y - yy - 2 : yy);

			    // Double-check bounds
			    if (zz < 0 || zz >= Z || xx < 0 || xx >= X || yy < 0 || yy >= Y) continue;

			    int idx_img = zz * (X * Y) + xx * Y + yy;
			    float val = image[idx_img];
			    sample_vals[count] = val;
				//raw_vals[count] = raw[idx_img];
				raw_vals[count] = (float)raw[idx_img];

			    if (val < min_val) min_val = val;

			    // Calculate Gaussian weight
			    float norm = expf(-(dz * dz / (2 * sigmaZ * sigmaZ) +
			                      dx * dx / (2 * sigmaXY * sigmaXY) +
			                      dy * dy / (2 * sigmaXY * sigmaXY)));
			    norm_G[count] = norm;

			    // For weighted center calculation
				// geometric center
			    //sum_val += val;
			    //sum_z += dz * val;
			    //sum_x += dx * val;
			    //sum_y += dy * val;
				// intensity center
				sum_val += val - min_val;  // Subtract background from each value
				sum_z += (zz) * (val - min_val);  // Use actual coordinate (not offset)
				sum_x += (xx) * (val - min_val);  // and subtract background
				sum_y += (yy) * (val - min_val);

			    count++;
			}
		}
	}
	if (count == 0) return;  // Safety check

	// Step 2: Calculate mean and std of Gaussian weights
	float mean_G = 0.0f;
	float var_G = 0.0f;

	for (int i = 0; i < count; i++) {
		mean_G += norm_G[i];
	}
	mean_G /= count;

	for (int i = 0; i < count; i++) {
		float diff = norm_G[i] - mean_G;
		var_G += diff * diff;
	}
	var_G /= count;
	float std_G = sqrtf(var_G);

	// Normalize Gaussian weights
	for (int i = 0; i < count; i++) {
		norm_G[i] = (norm_G[i] - mean_G) / std_G;
	}

	// Step 3: Calculate mean and std of image values
	float mean_sample = 0.0f;
	float var_sample = 0.0f;
	float mean_raw = 0.0f;
	float var_raw = 0.0f;

	for (int i = 0; i < count; i++) {
		mean_sample += sample_vals[i];
		mean_raw += raw_vals[i];
	}
	mean_sample /= count;
	mean_raw /= count;

	for (int i = 0; i < count; i++) {
		float diff_sample = sample_vals[i] - mean_sample;
		float diff_raw = raw_vals[i] - mean_raw;
		var_sample += diff_sample * diff_sample;
		var_raw += diff_raw * diff_raw;
	}
	var_sample /= count;
	var_raw /= count;
	float std_sample = sqrtf(var_sample);
	float std_raw = sqrtf(var_raw);
	// Step 4: Calculate normalized cross-correlation
	float hn = 0.0f;  // Cross-correlation with image
	float a = 0.0f;   // Cross-correlation with raw
	
	for (int i = 0; i < count; i++) {
		float norm_sample = (sample_vals[i] - mean_sample) / std_sample;
		float norm_raw = (raw_vals[i] - mean_raw) / std_raw;
		hn += norm_sample * norm_G[i];
		a += norm_raw * norm_G[i];
	}
	hn /= count;
	a /= count;

	// Calculate center coordinates
	// geometric center
	//float center_z = (count > 0) ? z0 + sum_z / (float)count : z0;
	//float center_x = (count > 0) ? x0 + sum_x / (float)count : x0;
	//float center_y = (count > 0) ? y0 + sum_y / (float)count : y0;
	// intensity center
	float center_z = (sum_val > 0) ? sum_z / sum_val : z0;
	float center_x = (sum_val > 0) ? sum_x / sum_val : x0;
	float center_y = (sum_val > 0) ? sum_y / sum_val : y0;
	
	int center_idx = z0 * (X * Y) + x0 * Y + y0;
	// Output: [zc, xc, yc, background, a, habs, hn, h]
	output[idx * 8 + 0] = center_z;
	output[idx * 8 + 1] = center_x;
	output[idx * 8 + 2] = center_y;
	output[idx * 8 + 3] = min_val;  // background
	output[idx * 8 + 4] = a;		// Cross-correlation with raw
	output[idx * 8 + 5] = image[center_idx];
	output[idx * 8 + 6] = hn;	   // Cross-correlation with image
	output[idx * 8 + 7] = (float)raw[center_idx];
}
