#include "hip/hip_runtime.h"
// riemutils/csrc/hyper_butterfly_cuda.cu

#include <torch/extension.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <cmath>
#include <vector>
#include "common_defs.h"
#include "hyper_butterfly.h"
#include "maps.h"
#include "butterfly.h"

using riemutils::butterfly_layer_cuda;
using riemutils::butterfly_layer_backward_cuda;

#define CHECK_CUDA_CONTIGUOUS(x)                                    \
  TORCH_CHECK((x).device().is_cuda(), #x " must be CUDA tensor");   \
  TORCH_CHECK((x).is_contiguous(), #x " must be contiguous")
#define CUDA_CHECK(err)                                             \
  do {                                                              \
    auto e = (err);                                                 \
    TORCH_CHECK(e == hipSuccess, "CUDA error: ",                   \
                hipGetErrorString(e));                             \
  } while (0)

// atanh 헬퍼 (clamp 포함)
__device__ __forceinline__ float atanh_device(float x) {
  x = fminf(fmaxf(x, -1.0f + 1e-6f), 1.0f - 1e-6f);
  return 0.5f * logf((1.0f + x) / (1.0f - x));
}

// ─────────────────────────────────────────────────────────────────────────────
// 1) 로그 맵 forward 커널
//    y = atanh(√c‖x‖)/(√c‖x‖) * x
// ─────────────────────────────────────────────────────────────────────────────
template <typename scalar_t>
__global__ void log_map_origin_kernel(
    const scalar_t* __restrict__ x,
    scalar_t*       __restrict__ out,
    float c, int B, int D) {

  extern __shared__ float sdata[];
  float* s_norm2 = sdata;  // shared[0]

  // ─── 반드시 block 당 0으로 초기화 ─────────────────────
  if (threadIdx.x == 0) {
    s_norm2[0] = 0.f;
  }
  __syncthreads();
  // ────────────────────────────────────────────────────

  int bid = blockIdx.x, tid = threadIdx.x, stride = blockDim.x;
  const scalar_t* xb = x + bid*D;
  scalar_t*       yb = out + bid*D;

  // 1) ||x||^2 reduction
  float local = 0.f;
  for (int i = tid; i < D; i += stride) {
    float v = xb[i];
    local += v*v;
  }
  // warp‐reduce
  for (int off = warpSize/2; off > 0; off >>= 1) {
    local += __shfl_down_sync(0xffffffff, local, off);
  }
  if ((tid & (warpSize-1)) == 0) {
    atomicAdd(s_norm2, local);
  }
  __syncthreads();

  // 2) clamp & factor
  if (tid == 0) {
    s_norm2[0] = fmaxf(s_norm2[0], EPS);
  }
  __syncthreads();
  float norm = sqrtf(s_norm2[0]);
  float u    = sqrtf(c)*norm;
  u = fminf(fmaxf(u, 1e-6f), 0.999999f);
  float factor = atanh_device(u)/(u + 1e-6f);

  // 3) output
  for (int i = tid; i < D; i += stride) {
    yb[i] = factor * xb[i];
  }
}

// ─────────────────────────────────────────────────────────────────────────────
// 2) exp 맵 forward 커널
//    y = tanh(√c‖v‖)/(√c‖v‖) * v
// ─────────────────────────────────────────────────────────────────────────────
template <typename scalar_t>
__global__ void exp_map_origin_kernel(
    const scalar_t* __restrict__ v,
    scalar_t*       __restrict__ out,
    float c, int B, int D) {

  extern __shared__ float sdata[];
  float* s_norm2 = sdata;  // shared[0]

  // ─── 반드시 block 당 0으로 초기화 ─────────────────────
  if (threadIdx.x == 0) {
    s_norm2[0] = 0.f;
  }
  __syncthreads();
  // ────────────────────────────────────────────────────

  int bid = blockIdx.x, tid = threadIdx.x, stride = blockDim.x;
  const scalar_t* vb = v + bid*D;
  scalar_t*       yb = out + bid*D;

  // 1) ||v||^2 reduction
  float local = 0.f;
  for (int i = tid; i < D; i += stride) {
    float w = vb[i];
    local += w*w;
  }
  for (int off = warpSize/2; off > 0; off >>= 1) {
    local += __shfl_down_sync(0xffffffff, local, off);
  }
  if ((tid & (warpSize-1)) == 0) {
    atomicAdd(s_norm2, local);
  }
  __syncthreads();

  if (tid == 0) {
    s_norm2[0] = fmaxf(s_norm2[0], EPS);
  }
  __syncthreads();

  float norm = sqrtf(s_norm2[0]);
  float u    = sqrtf(c)*norm;
  u = fminf(fmaxf(u, 1e-6f), 10.0f);
  float tanhu = tanhf(u);
  float factor = tanhu/(u + 1e-3f);

  // 2) output
  for (int i = tid; i < D; i += stride) {
    yb[i] = factor * vb[i];
  }
}
// ─────────────────────────────────────────────────────────────────────────────
// 3) Butterfly 레이어 (forward)
// ─────────────────────────────────────────────────────────────────────────────
template <typename scalar_t>
__global__ void butterfly_layer_kernel(
    const scalar_t* __restrict__ input,
    scalar_t*       __restrict__ output,
    const scalar_t* __restrict__ params,
    int B, int D, int layer_idx) {

  int idx    = blockIdx.x*blockDim.x + threadIdx.x;
  int stride = blockDim.x*gridDim.x;
  int bs     = 1 << layer_idx;
  int nb     = D / (2*bs);

  while(idx < B*D) {
    int b = idx / D, f = idx % D;
    int blk = (f/(2*bs)) % nb,
        loc = f % (2*bs),
        off = loc % bs;
    bool high = loc >= bs;
    int pi = blk*2;
    float a  = params[pi+0],
          bb = params[pi+1];
    int base = b*D + blk*2*bs;
    float x1 = input[base + off],
          x2 = input[base + off + bs];
    output[idx] = high
      ? (-bb*x1 + a*x2)
      : ( a*x1 + bb*x2 );
    idx += stride;
  }
}

template <typename scalar_t>
__global__ void butterfly_layer_backward_kernel(
    const scalar_t* __restrict__ grad_out,
    const scalar_t* __restrict__ input,
    scalar_t*       __restrict__ grad_in,
    const scalar_t* __restrict__ params,
    scalar_t*       __restrict__ grad_params,
    int B, int D, int layer_idx) 
{
  int idx = blockIdx.x*blockDim.x + threadIdx.x;
  int stride = blockDim.x*gridDim.x;
  int bs = 1<<layer_idx, nb = D/(2*bs);

  while(idx < B*D) {
    int b = idx/D, f = idx%D;
    int blk = (f/(2*bs))%nb, loc = f%(2*bs), off = loc%bs;
    bool high = loc>=bs;
    int pi = blk*2;
    float a = params[pi+0], bb = params[pi+1];
    int base = b*D + blk*2*bs;
    float x1 = input[base+off], x2 = input[base+off+bs];
    float gout = grad_out[idx];

    if(!high) {
      // y = a*x1 + b*x2
      atomicAdd(&grad_in[base+off  ],  a*gout);
      atomicAdd(&grad_in[base+off+bs],  bb*gout);
      atomicAdd(&grad_params[pi+0], x1*gout);
      atomicAdd(&grad_params[pi+1], x2*gout);
    } else {
      // y = -b*x1 + a*x2
      atomicAdd(&grad_in[base+off  ], -bb*gout);
      atomicAdd(&grad_in[base+off+bs],  a*gout);
      atomicAdd(&grad_params[pi+0],  x2*gout);
      atomicAdd(&grad_params[pi+1], -x1*gout);
    }
    idx += stride;
  }
}

// ─────────────────────────────────────────────────────────────────────────────
// 3) exp_map backward 커널
// ─────────────────────────────────────────────────────────────────────────────
template <typename scalar_t>
__global__ void exp_map_backward_kernel(
    const scalar_t* __restrict__ v,
    const scalar_t* __restrict__ grad_y,
    scalar_t*       __restrict__ grad_v,
    float c, int B, int D) {

  extern __shared__ float sdata[];
  float* s_v2 = sdata;      // [0]
  float* s_vg = sdata + 1;  // [1]

  // ─── 반드시 block 당 0으로 초기화 ─────────────────────
  if (threadIdx.x == 0) {
    s_v2[0] = 0.f;
    s_vg[0] = 0.f;
  }
  __syncthreads();
  // ────────────────────────────────────────────────────

  int bid = blockIdx.x, tid = threadIdx.x, stride = blockDim.x;
  const scalar_t* vb = v + bid*D;
  const scalar_t* gy = grad_y + bid*D;

  // 1) ||v||^2, v·grad_y reduction
  float local_v2 = 0.f, local_vg = 0.f;
  for (int i = tid; i < D; i += stride) {
    float vv = vb[i], gyv = gy[i];
    local_v2 += vv*vv;
    local_vg += vv*gyv;
  }
  for (int off = warpSize/2; off > 0; off >>= 1) {
    local_v2 += __shfl_down_sync(0xffffffff, local_v2, off);
    local_vg += __shfl_down_sync(0xffffffff, local_vg, off);
  }
  if ((tid & (warpSize-1)) == 0) {
    atomicAdd(s_v2, local_v2);
    atomicAdd(s_vg, local_vg);
  }
  __syncthreads();

  if (threadIdx.x == 0) {
    s_v2[0] = fmaxf(s_v2[0], EPS);
  }
  __syncthreads();

  float norm = sqrtf(s_v2[0]);
  float u    = sqrtf(c)*norm;
  u = fminf(fmaxf(u, 1e-6f), 10.0f);
  float tanhu = tanhf(u);
  float sech2 = 1.0f - tanhu*tanhu;
  float factor = tanhu/(u + 1e-3f);

  // d factor / d norm
  float df_du = (u*sech2 - tanhu)/(u*u);
  float df_dn = df_du*sqrtf(c);
  float vdotgy= s_vg[0];

  // 2) per-dim gradient
  for (int i = tid; i < D; i += stride) {
    float vi  = vb[i], gyi = gy[i];
    grad_v[bid*D + i] = factor*gyi + (vi/norm)*(df_dn*vdotgy);
  }
}

// ─────────────────────────────────────────────────────────────────────────────
// 4) log_map backward 커널
// ─────────────────────────────────────────────────────────────────────────────
template <typename scalar_t>
__global__ void log_map_backward_kernel(
    const scalar_t* __restrict__ x,
    const scalar_t* __restrict__ grad_u,
    scalar_t*       __restrict__ grad_x,
    float c, int B, int D) {

  extern __shared__ float sdata[];
  float* s_x2 = sdata;      // [0]
  float* s_xu = sdata + 1;  // [1]

  // ─── 반드시 block 당 0으로 초기화 ─────────────────────
  if (threadIdx.x == 0) {
    s_x2[0] = 0.f;
    s_xu[0] = 0.f;
  }
  __syncthreads();
  // ────────────────────────────────────────────────────

  int bid = blockIdx.x, tid = threadIdx.x, stride = blockDim.x;
  const scalar_t* xb = x     + bid*D;
  const scalar_t* gu = grad_u+ bid*D;

  // 1) ||x||^2, x·grad_u reduction
  float local_x2 = 0.f, local_xu = 0.f;
  for (int i = tid; i < D; i += stride) {
    float xi  = xb[i], gui = gu[i];
    local_x2 += xi*xi;
    local_xu += xi*gui;
  }
  for (int off = warpSize/2; off > 0; off >>= 1) {
    local_x2 += __shfl_down_sync(0xffffffff, local_x2, off);
    local_xu += __shfl_down_sync(0xffffffff, local_xu, off);
  }
  if ((tid & (warpSize-1)) == 0) {
    atomicAdd(s_x2, local_x2);
    atomicAdd(s_xu, local_xu);
  }
  __syncthreads();

  if (threadIdx.x == 0) {
    s_x2[0] = fmaxf(s_x2[0], EPS);
  }
  __syncthreads();

  float norm = sqrtf(s_x2[0]);
  float u    = sqrtf(c)*norm;
  u = fminf(fmaxf(u, 1e-6f), 0.999999f);
  
  // atanh(u) 계산
  float atanhu = 0.5f * logf((1.0f + u) / (1.0f - u));
  float factor = atanhu/(u + 1e-6f);

  float sech2 = 1.0f - u*u;
  float df_du = (u*sech2 - atanhu)/(u*u);
  float df_dn = df_du*sqrtf(c);
  float xdotg = s_xu[0];

  // 2) per-dim gradient
  for (int i = tid; i < D; i += stride) {
    float xi   = xb[i], guv = gu[i];
    grad_x[bid*D + i] = factor*guv + (xi/norm)*(df_dn*xdotg);
  }
}


// 2. log_map_origin_cuda 및 exp_map_origin_cuda 구현
torch::Tensor log_map_origin_cuda(torch::Tensor x, float c) {
  CHECK_CUDA_CONTIGUOUS(x);
  int B = x.size(0), D = x.size(1);
  auto out = torch::empty_like(x);
  int threads = std::min(D, 1024);
  int shbytes = sizeof(float);
  
  AT_DISPATCH_FLOATING_TYPES(x.scalar_type(), "log_map_origin_cuda", [&]{
    log_map_origin_kernel<scalar_t><<<B, threads, shbytes>>>(
      x.data_ptr<scalar_t>(),
      out.data_ptr<scalar_t>(),
      c, B, D);
  });
  
  CUDA_CHECK(hipGetLastError());
  return out;
}

torch::Tensor exp_map_origin_cuda(torch::Tensor v, float c) {
  CHECK_CUDA_CONTIGUOUS(v);
  int B = v.size(0), D = v.size(1);
  auto out = torch::empty_like(v);
  int threads = std::min(D, 1024);
  int shbytes = sizeof(float);
  
  AT_DISPATCH_FLOATING_TYPES(v.scalar_type(), "exp_map_origin_cuda", [&]{
    exp_map_origin_kernel<scalar_t><<<B, threads, shbytes>>>(
      v.data_ptr<scalar_t>(),
      out.data_ptr<scalar_t>(),
      c, B, D);
  });
  
  CUDA_CHECK(hipGetLastError());
  return out;
}

std::vector<torch::Tensor> hyper_butterfly_cuda(
    torch::Tensor x,
    torch::Tensor params,
    torch::Tensor unused,
    float c,
    int L)
{
  CHECK_CUDA_CONTIGUOUS(x);
  CHECK_CUDA_CONTIGUOUS(params);

  int B = x.size(0), D = x.size(1);
  int D_padded = next_pow2(D);
  
  // Step 1: Pad input if needed
  torch::Tensor x_padded;
  if (D_padded > D) {
    x_padded = torch::zeros({B, D_padded}, x.options());
    x_padded.narrow(1, 0, D).copy_(x);
  } else {
    x_padded = x;
  }

  // Step 2: Log map
  torch::Tensor u = log_map_origin_cuda(x_padded, c);
  
  // Step 3: Apply butterfly transforms
  torch::Tensor v = u.clone();
  
  for (int l = 0; l < L; l++) {
    int layer_idx = l % int(std::log2(D_padded));
    v = butterfly_layer_cuda(v, params, layer_idx, B, D_padded);
  }
  
  // Step 4: Exp map
  torch::Tensor y_padded = exp_map_origin_cuda(v, c);
  
  // Step 5: Slice to original dimension if needed
  torch::Tensor y = (D_padded > D) ? y_padded.narrow(1, 0, D) : y_padded;
  
  return {y, u, v};
}

std::vector<torch::Tensor> hyper_butterfly_backward_cuda(
    torch::Tensor grad_y,
    torch::Tensor x,
    torch::Tensor params,
    float c,
    int L)
{
  CHECK_CUDA_CONTIGUOUS(grad_y);
  CHECK_CUDA_CONTIGUOUS(x);
  CHECK_CUDA_CONTIGUOUS(params);

  int B = x.size(0), D = x.size(1);
  int D_padded = next_pow2(D);
  
  // Step 1: Pad input if needed
  torch::Tensor x_padded, grad_y_padded;
  if (D_padded > D) {
    x_padded = torch::zeros({B, D_padded}, x.options());
    x_padded.narrow(1, 0, D).copy_(x);
    
    grad_y_padded = torch::zeros({B, D_padded}, grad_y.options());
    grad_y_padded.narrow(1, 0, D).copy_(grad_y);
  } else {
    x_padded = x;
    grad_y_padded = grad_y;
  }

  // Step 2: Forward pass to get intermediate results
  torch::Tensor u = log_map_origin_cuda(x_padded, c);
  
  // Apply butterfly transforms (forward)
  std::vector<torch::Tensor> intermediates;
  intermediates.push_back(u);
  
  torch::Tensor v = u.clone();
  
  for (int l = 0; l < L; l++) {
    int layer_idx = l % int(std::log2(D_padded));
    v = butterfly_layer_cuda(v, params, layer_idx, B, D_padded);
    intermediates.push_back(v);
  }
  
  // Final forward result
  torch::Tensor y_padded = exp_map_origin_cuda(v, c);
  
  // Step 3: Backward pass
  // Starting with grad_out at exp_map
  torch::Tensor grad_v = torch::zeros_like(v);
  int threads = std::min(D_padded, 1024);
  int shbytes = 2 * sizeof(float);
  
  // Backward through exp_map
  AT_DISPATCH_FLOATING_TYPES(v.scalar_type(), "exp_map_backward_cuda", [&]{
    exp_map_backward_kernel<scalar_t><<<B, threads, shbytes>>>(
      v.data_ptr<scalar_t>(),
      grad_y_padded.data_ptr<scalar_t>(),
      grad_v.data_ptr<scalar_t>(),
      c, B, D_padded);
  });
  
  // Backward through butterfly layers
  auto grad_params = torch::zeros_like(params);
  auto grad_u = torch::zeros_like(u);
  
  // Final layer's grad_out is grad_v
  torch::Tensor grad_curr = grad_v;
  
  // Backward through butterfly layers (in reverse order)
  for (int l = L-1; l >= 0; l--) {
    int layer_idx = l % int(std::log2(D_padded));
    torch::Tensor input = intermediates[l];
    
    // Butterfly backward
    auto result = butterfly_layer_backward_cuda(
      grad_curr, input, params, layer_idx);
    
    torch::Tensor grad_input = result[0];
    torch::Tensor layer_grad_params = result[1];
    
    // Accumulate parameter gradients
    int p_offset = 0;
    for (int i = 0; i < layer_idx; i++) {
      int block_size = 1 << i;
      p_offset += 2 * (D_padded / (2 * block_size));
    }
    int p_size = 2 * (D_padded / (2 * (1 << layer_idx)));
    grad_params.narrow(0, p_offset, p_size).add_(layer_grad_params.narrow(0, p_offset, p_size));
    
    // Update grad for next layer
    grad_curr = grad_input;
  }
  
  // Set grad_u for log_map backward
  grad_u = grad_curr;
  
  // Backward through log_map
  torch::Tensor grad_x_padded = torch::zeros_like(x_padded);
  AT_DISPATCH_FLOATING_TYPES(x_padded.scalar_type(), "log_map_backward_cuda", [&]{
    log_map_backward_kernel<scalar_t><<<B, threads, shbytes>>>(
      x_padded.data_ptr<scalar_t>(),
      grad_u.data_ptr<scalar_t>(),
      grad_x_padded.data_ptr<scalar_t>(),
      c, B, D_padded);
  });
  
  // Get gradients for original dimensions
  torch::Tensor grad_x = (D_padded > D) ? grad_x_padded.narrow(1, 0, D) : grad_x_padded;
  
  return {grad_x, grad_params};
}

namespace riemutils {

torch::Tensor butterfly_layer_cuda(
    torch::Tensor input,
    torch::Tensor params,
    int layer_idx,
    int batch_size,
    int dim) {
    
    auto output = torch::empty_like(input);
    dim3 grid(std::min((batch_size * dim + 511) / 512, 1024));
    dim3 block(512);
    
    AT_DISPATCH_FLOATING_TYPES(input.scalar_type(), "butterfly_layer_cuda", ([&] {
        butterfly_layer_kernel<scalar_t><<<grid, block>>>(
            input.data_ptr<scalar_t>(),
            output.data_ptr<scalar_t>(),
            params.data_ptr<scalar_t>(),
            batch_size, dim, layer_idx);
    }));
    
    CUDA_CHECK(hipGetLastError());
    return output;
}

std::vector<torch::Tensor> butterfly_layer_backward_cuda(
    torch::Tensor grad_out,
    torch::Tensor input,
    torch::Tensor params,
    int layer_idx) {
    
    auto grad_input = torch::zeros_like(input);
    auto grad_params = torch::zeros_like(params);
    int batch_size = input.size(0);
    int dim = input.size(1);
    
    dim3 grid(std::min((batch_size * dim + 511) / 512, 1024));
    dim3 block(512);
    
    AT_DISPATCH_FLOATING_TYPES(input.scalar_type(), "butterfly_layer_backward_cuda", ([&] {
        butterfly_layer_backward_kernel<scalar_t><<<grid, block>>>(
            grad_out.data_ptr<scalar_t>(),
            input.data_ptr<scalar_t>(),
            grad_input.data_ptr<scalar_t>(),
            params.data_ptr<scalar_t>(),
            grad_params.data_ptr<scalar_t>(),
            batch_size, dim, layer_idx);
    }));
    
    CUDA_CHECK(hipGetLastError());
    return {grad_input, grad_params};
}

} // namespace riemutils

